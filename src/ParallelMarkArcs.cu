#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>

#include <time.h>

#include <InteractionArcs.h>

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__device__ int anchor_length(int start, int end) {
  if (start == 0 && end == 0)
    return 0;
  return end - start + 1;
}

__device__ bool anchor_contains(int start, int end, int pos) {
  return pos >= start && pos <= end;
}

__global__ void kernel(int *__restrict__ anchors_starts,
                       int *__restrict__ anchors_ends,
                       int *__restrict__ raw_arcs_starts,
                       int *__restrict__ raw_arcs_ends, int *output_starts,
                       int *output_ends, const int anchors_count,
                       const int arcs_count) {
  int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  if (threadIndex >= arcs_count)
    return;

  for (int i = threadIndex; i < arcs_count; i += stride) {
    int start = -1, end = -1;

    for (int j = 0; j < anchors_count; ++j) {
      if (anchor_length(anchors_starts[j], anchors_ends[j]) > 1) {
        if (anchor_contains(anchors_starts[j], anchors_ends[j],
                            raw_arcs_starts[i]))
          output_starts[i] = j;
        if (anchor_contains(anchors_starts[j], anchors_ends[j],
                            raw_arcs_ends[i]))
          output_ends[i] = j;
        if (start != -1 && end != -1)
          return;
      }
    }
  }
}

// we have anchors[], a sorted list of anchors, and raw_arcs[], a list of arcs
// (using genomic positions) we want to fill arc[] so that it contains a list of
// arcs with values referring to indices of 'anchors', and not genomic positions
void InteractionArcs::parallelMarkArcs(bool ignore_missing) {
  int threads = 1024;
  int blocks = 256;

  // for every chromosome:
  // go through all raw arcs, find the corresponding anchor, create new arc
  // (anchor based) and add it to list

  int cnt = 0;
  int last_start = -1;
  std::unordered_map<int, std::vector<InteractionArc>> tmp_arcs;

  // normally we want to print warning about mismatching arcs (ones for which
  // anchors are missing), but if we have selected a region (either by providing
  // specific region or by limiting region in debug mode) then we want to
  // surpress them (because we can expect that there are going to be some
  // mismatches anyway)
  // TODO: we do filtering when reading arcs, so maybe there should be no
  // mismatches? bool mismatched_arcs_as_errors = ignore_missing ||
  // (selected_region.end == 0);

  for (string chr : chrs) {
    printf(" %s...\n", chr.c_str());

    arcs[chr].clear(); // we may run markArcs() multiple times. make sure we
                       // won't duplicate arcs

    std::sort(raw_arcs[chr].begin(), raw_arcs[chr].end());

    cnt = raw_arcs[chr].size();

    std::vector<Anchor> current_anchors = anchors[chr];
    std::vector<InteractionArc> current_raw_arcs = raw_arcs[chr];

    thrust::host_vector<int> h_anchors_start(current_anchors.size());
    thrust::host_vector<int> h_anchors_end(current_anchors.size());
    thrust::host_vector<int> h_raw_arcs_start(cnt);
    thrust::host_vector<int> h_raw_arcs_end(cnt);
    thrust::host_vector<int> h_outputs_start;
    thrust::host_vector<int> h_outputs_end;

    thrust::device_vector<int> d_anchors_start;
    thrust::device_vector<int> d_anchors_end;
    thrust::device_vector<int> d_raw_arcs_start;
    thrust::device_vector<int> d_raw_arcs_end;
    thrust::device_vector<int> d_outputs_start(cnt);
    thrust::device_vector<int> d_outputs_end(cnt);

    thrust::fill(d_outputs_start.begin(), d_outputs_start.end(), -1);
    thrust::fill(d_outputs_end.begin(), d_outputs_end.end(), -1);

    for (int i = 0; i < anchors_cnt[chr]; ++i) {
      h_anchors_start[i] = current_anchors[i].start;
      h_anchors_end[i] = current_anchors[i].end;
    }

    for (int i = 0; i < cnt; ++i) {
      h_raw_arcs_start[i] = current_raw_arcs[i].start;
      h_raw_arcs_end[i] = current_raw_arcs[i].end;
    }

    d_anchors_start = h_anchors_start;
    d_anchors_end = h_anchors_end;
    d_raw_arcs_start = h_raw_arcs_start;
    d_raw_arcs_end = h_raw_arcs_end;

    kernel<<<blocks, threads>>>(
        thrust::raw_pointer_cast(d_anchors_start.data()),
        thrust::raw_pointer_cast(d_anchors_end.data()),
        thrust::raw_pointer_cast(d_raw_arcs_start.data()),
        thrust::raw_pointer_cast(d_raw_arcs_end.data()),
        thrust::raw_pointer_cast(d_outputs_start.data()),
        thrust::raw_pointer_cast(d_outputs_end.data()), anchors_cnt[chr], cnt);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    h_outputs_start = d_outputs_start;
    h_outputs_end = d_outputs_end;

    // we need i==cnt to process the remaining arcs
    for (int i = 0; i <= cnt; ++i) { // for every arc

      int st = -1, end = -1;

      if (i < cnt) {
        st = h_outputs_start[i];
        end = h_outputs_end[i];

        if ((st == -1 || end == -1)) {
          printf("! error: non-matching arc\n");
          raw_arcs[chr][i].print();
          continue;
        }

        if (st == end) {
          // printf("st==end %d %d\n", st, end);
          // raw_arcs[chr][i].print();
          continue; // ignore looping arcs
        }
      }

      if (st != last_start || i == cnt) {

        // add all cached arcs
        for (auto el : tmp_arcs) {
          // here 'el.second' is a vector with arcs having common start and end
          // if there is only one arc, then we can simply add it to the list
          if (el.second.size() == 1) {
            arcs[chr].push_back(el.second[0]);
          } else {

            // sort arcs (so that they are ordered by factor)
            std::sort(el.second.begin(), el.second.end());

            // check how many different factors there are
            bool multiple_factors = false;
            for (size_t j = 1; j < el.second.size(); ++j)
              if (el.second[j].factor != el.second[j - 1].factor)
                multiple_factors = true;

            int total_score = 0;
            int factor_score = 0;
            int first_of_factor = 0;
            for (size_t j = 0; j <= el.second.size(); ++j) {

              // if factor is changing update the arc
              if (j == el.second.size() ||
                  (j > 0 && el.second[j].factor != el.second[j - 1].factor)) {
                el.second[first_of_factor].score = factor_score;
                el.second[first_of_factor].eff_score =
                    multiple_factors ? 0 : factor_score;
                arcs[chr].push_back(el.second[first_of_factor]);

                first_of_factor = j;
                total_score += factor_score;
                factor_score = 0;
              }

              if (j < el.second.size())
                factor_score += el.second[j].score;
            }

            // if we had multiple factors create a single, summary arc
            if (multiple_factors) {
              InteractionArc arc(el.second[0].start, el.first, 0, -1);
              arc.eff_score = total_score;
              arcs[chr].push_back(arc);
            }
          }

          el.second.clear();
        }
        tmp_arcs.clear();
        last_start = st;
      }

      // we will gather all arcs starting at anchor 'st', and process them
      // together (we do that because we need to merge arcs between the same
      // anchors but with different factors)
      InteractionArc arc(st, end, raw_arcs[chr][i].score,
                         raw_arcs[chr][i].factor);
      arc.genomic_start = raw_arcs[chr][i].start;
      arc.genomic_end = raw_arcs[chr][i].end;
      tmp_arcs[end].push_back(arc);
    }

    arcs_cnt[chr] = arcs[chr].size(); // update count
    tmp_arcs.clear();
  }
}
